#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define REPETITIONS 1
/*
 * Each SM has 4 processing blocks, and each processing block contains 8 FP64
 * cores capable of performing FMA (Fused Multiply-Add)
 */
#define THREADS_PER_BLOCK 8 * 4

__global__ void matrixMultiplicationKernel(const double *A, const double *B,
                                           double *C, int n) {
  // Calculate thread IDs
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < n && col < n) {
    double sum = 0.0;
    for (int k = 0; k < n; ++k) {
      sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
  }
}

void matrixMultiplicationCUDA(const double *A, const double *B, double *C,
                              int n) {
  // Allocate memory on GPU
  double *d_A, *d_B, *d_C;
  size_t size = n * n * sizeof(double);

  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  // Copy data from host (CPU) to device (GPU)
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  // Define grid and block configuration
  dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  dim3 numBlocks(n / threadsPerBlock.x, n / threadsPerBlock.y);

  // Execute kernel
  matrixMultiplicationKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, n);

  // Synchronize to ensure kernel completion
  hipDeviceSynchronize();

  // Copy results back from device to host
  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  // Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}


int main(int argc, char **argv) {
  if (argc != 2) {
    fprintf(stderr, "Usage: %s <matrix_size>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);
  if (N <= 0) {
    fprintf(stderr, "Matrix size must be a positive integer.\n");
    return EXIT_FAILURE;
  }

  // Allocate memory for matrices on the host
  size_t size = N * N * sizeof(double);
  double *A = (double *)malloc(size);
  double *B = (double *)malloc(size);
  double *C = (double *)malloc(size);

  // data initializing
  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      A[i * N + j] = (double)(rand()) / RAND_MAX * 10;
      B[i * N + j] = (double)(rand()) / RAND_MAX * 10;
      C[i * N + j] = 0.0;
    }
  }

  // Time measurement
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  for (int i = 0; i < REPETITIONS; i++) {
    matrixMultiplicationCUDA(A, B, C, N);
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);


  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  double elapsed_time = ((double)(milliseconds));

  printf("Time taken: %f milliseconds\n", elapsed_time);

  free(A);
  free(B);
  free(C);

  return EXIT_SUCCESS;
}